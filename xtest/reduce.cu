#include "common/xtestrunner.h"
#include <xcbb/xcbb.h>


uint SerialReduce(const std::vector<uint>& in)
{
    uint reduce = 0;
    for (int i = 0;  i < in.size(); ++i) {
        reduce += in[i];
    }
    return reduce;
}


TEST_F(CudaTest, ParallelReduceSingle)
{
    std::srand(time(0));
    const int numElements = 16384111;

    std::vector<uint> data(numElements);

    for (int i = 0; i < numElements; ++i)
        data[i] = 1; //rand() % 100;

    // Push scanned array to device
    uint* d_data;
    checkCudaErrors(hipMalloc((void**) &d_data,    sizeof(uint) * numElements));
    checkCudaErrors(hipMemcpy(d_data, data.data(), sizeof(uint) * numElements, hipMemcpyHostToDevice));

    // Reference serial version
    uint serialReduce = SerialReduce(data);

    // Initialize reduce storage
    ReduceStorage<uint> storage(numElements);
    storage.InitDeviceStorage(d_data);

    // Create reduce enactor
    ReduceEnactor<uint> reduce(numElements);

    // Perform scan algorithm
    CudaDeviceTimer timer;
    timer.Start();
    uint deviceResult = reduce.Enact(storage);
    timer.Stop();

    checkCudaErrors(hipFree(d_data));

    EQUAL(serialReduce, deviceResult);
    printf("Results:    %d  %d\n", serialReduce, deviceResult);
    printf("Problem:    %d\n", numElements);
    printf("Time:       %.3f [ms]\n", timer.ElapsedTime());
}



TEST_F(CudaTest, ParallelReduceMany)
{
    const int n[] = { 128*512,
                      128*512*4,
                      128*512*100,
                      128*512 + 1*128,
                      128*512 + 2*128,
                      128*512 + 3*128,
                      128*512 + 33,
                      128*512 + 127,
                      128*512 + 128 + 45,
                      128*512 + 512 + 17,
                      128*512 + 512 + 2*128,
                      128*512 + 60*512 + 3*128 + 65};

    std::srand(time(0));
    int numElements = 0;

    for (int isize = 0; isize < sizeof(n)/sizeof(n[0]); isize++)
    {
        numElements = n[isize];

        std::vector<uint> data(numElements);

        for (int i = 0; i < numElements; ++i)
            data[i] = rand() % 100;

        // Push scanned array to device
        uint* d_data;
        checkCudaErrors(hipMalloc((void**) &d_data,    sizeof(uint) * numElements));
        checkCudaErrors(hipMemcpy(d_data, data.data(), sizeof(uint) * numElements, hipMemcpyHostToDevice));

        // Reference serial version
        uint serialReduce = SerialReduce(data);

        // Initialize reduce storage
        ReduceStorage<uint> storage(numElements);
        storage.InitDeviceStorage(d_data);

        // Create reduce enactor
        ReduceEnactor<uint> reduce(numElements);

        // Perform scan algorithm
        CudaDeviceTimer timer;
        timer.Start();
        uint deviceResult = reduce.Enact(storage);
        timer.Stop();

        checkCudaErrors(hipFree(d_data));

        EQUAL(serialReduce, deviceResult);
        printf("====================================================================================\n");
        printf("Results:    %d  %d\n", serialReduce, deviceResult);
        printf("Problem:    %d\n", numElements);
        printf("Time:       %.3f [ms]\n", timer.ElapsedTime());
        printf("====================================================================================\n");
    }
}


int main(int argc, char **argv)
{
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
