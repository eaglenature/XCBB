#include "common/xtestrunner.h"
#include "common/xtimer.h"
#include <xcbb/xcbb.h>


void SerialExclusiveScan(std::vector<uint>& out, const std::vector<uint>& in)
{
    uint sum = 0;
    for (int i = 0;  i < in.size(); ++i) {
        uint x = in[i];
        out[i] = sum;
        sum += x;
    }
}


TEST_F(CudaTest, ParallelExclusiveScanSingle)
{
    std::srand(time(0));
    const int numElements = 16384111;

    std::vector<uint> data(numElements);
    std::vector<uint> result(numElements);

    for (int i = 0; i < numElements; ++i)
        data[i] = rand() % 100;

    // Push scanned array to device
    uint* d_data;
    checkCudaErrors(hipMalloc((void**) &d_data,    sizeof(uint) * numElements));
    checkCudaErrors(hipMemcpy(d_data, data.data(), sizeof(uint) * numElements, hipMemcpyHostToDevice));

    // Reference serial version
    SerialExclusiveScan(data, data);

    // Initialize scan storage
    ExclusiveScanStorage<uint> storage(numElements);
    storage.InitDeviceStorage(d_data);

    // Create scan enactor
    ExclusiveScanEnactor<uint> scanner(numElements);

    // Perform scan algorithm
    CudaDeviceTimer timer;
    timer.Start();
    scanner.Enact(storage);
    timer.Stop();

    // Get scanned array back to host
    checkCudaErrors(hipMemcpy(result.data(), d_data, sizeof(uint) * numElements, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_data));

    EQUAL_RANGES(data, result);
    printf("Problem:    %d\n", numElements);
    printf("Time:       %.3f [ms]\n", timer.ElapsedTime());
}


TEST_F(CudaTest, ParallelExclusiveScanMany)
{
    const int n[] = { 128*512,
                      128*512*4,
                      128*512*100,
                      128*512 + 1*128,
                      128*512 + 2*128,
                      128*512 + 3*128,
                      128*512 + 33,
                      128*512 + 127,
                      128*512 + 128 + 45,
                      128*512 + 512 + 17,
                      128*512 + 512 + 2*128,
                      128*512 + 60*512 + 3*128 + 65};

    std::srand(time(0));
    int numElements = 0;

    for (int isize = 0; isize < sizeof(n)/sizeof(n[0]); isize++)
    {
        numElements = n[isize];

        std::vector<uint> data(numElements);
        std::vector<uint> result(numElements);

        for (int i = 0; i < numElements; ++i)
            data[i] = rand() % 100;

        // Push scanned array to device
        uint* d_data;
        checkCudaErrors(hipMalloc((void**) &d_data,    sizeof(uint) * numElements));
        checkCudaErrors(hipMemcpy(d_data, data.data(), sizeof(uint) * numElements, hipMemcpyHostToDevice));

        // Reference serial version
        SerialExclusiveScan(data, data);

        // Initialize scan storage
        ExclusiveScanStorage<uint> storage(numElements);
        storage.InitDeviceStorage(d_data);

        // Create scan enactor
        ExclusiveScanEnactor<uint> scanner(numElements);

        // Perform scan algorithm
        CudaDeviceTimer timer;
        timer.Start();
        scanner.Enact(storage);
        timer.Stop();

        // Get scanned array back to host
        checkCudaErrors(hipMemcpy(result.data(), d_data, sizeof(uint) * numElements, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_data));

        EQUAL_RANGES(data, result);
        printf("====================================================================================\n");
        printf("Problem:    %d\n", numElements);
        printf("Time:       %.3f [ms]\n", timer.ElapsedTime());
        printf("====================================================================================\n");
    }
}


int main(int argc, char **argv)
{
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}

